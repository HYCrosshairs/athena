#include "hip/hip_runtime.h"
#include "CudaVector.cuh"

#include <stdexcept>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace ai::ml::neural;

template<typename T>
__global__ void vectorMultiplyBy(T* dataInput, double scalar, size_t size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        dataInput[tid] = dataInput[tid] * scalar;
    }       
}

template<typename T>
CudaVector<T>::CudaVector(size_t size, T* inputData) : size(size), hostData(inputData)
{
    hipMalloc((void **)&deviceData, size * sizeof(T));

    hipMemcpy(deviceData, hostData, size * sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
CudaVector<T>::~CudaVector()
{
    hipFree(deviceData);
}

template<typename T>
void CudaVector<T>::cudaConfigureKernelCall(size_t threadsPerBlock)
{
    this->threadsPerBlock = threadsPerBlock;
    this->numBlocks = (this->size + threadsPerBlock - 1) / threadsPerBlock;
}

template<typename T>
void CudaVector<T>::cudaKernelCall(double scalar)
{
    vectorMultiplyBy<<<this->numBlocks, this->threadsPerBlock>>>(deviceData, scalar, this->size);

    hipError_t error = hipGetLastError();
    
    if (error != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }

    hipMemcpy(hostData, deviceData, size * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void CudaVector<T>::cudaShowKernelCallResults()
{
    for (size_t i = 0; i < this->size; i++)
    {
        std::cout << hostData[i] << std::endl;
    }
    
}

template class CudaVector<double>;